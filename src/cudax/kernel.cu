#include "util/gl_wrapper.h" ///< for cuda_gl_interop
#include <cuda_gl_interop.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>

#include "cudax/kernel.h"
#include "cudax/CudaTimer.h"
#include "cudax/hip/hip_runtime_api.h" ///< SDK error checking
#include "cudax/CublasHelper.h"
#include "cudax/CudaHelper.h"
#include "cudax/KinectCamera.h"
#include "cudax/kernel_init.h"
#include "cudax/kernel_upload.h"
#include "cudax/kernel_debug.h"
#include "cudax/PixelIndexer.h"

#include "cudax/functors/IsSilhouette.h"
#include "cudax/functors/ComputeJacobianSilhouette.h"
#include "cudax/functors/ComputeJacobianData.h"

#include <iostream>
#include <fstream>
#include <string>

using namespace cudax;

struct absolute_value : public thrust::unary_function <float, float > {
	__host__ __device__
		float operator()(float x) const {
		return (x>=0) ? x : -x;
	}
};

void kernel_bind()
{
    if(cudax::sensor_depth)   CHECK_CUDA(hipBindTextureToArray(depth_tex, cudax::sensor_depth));
}

void kernel_unbind(){
    if(cudax::sensor_depth)   CHECK_CUDA(hipUnbindTexture(depth_tex));
    cudax::sensor_depth=NULL;
}

void kernel(float* eigen_JtJ, float* eigen_JtF, float & push_error, float & pull_error, bool eval_metric, bool reweight, int id, int iter, 
	int num_sensor_points, int num_rendered_points) {    
    
    // CUDA_TIMED_BLOCK(timer,"indexing constraints")

	int n_pull, n_push;

	if (settings->fit2D_enable) {
		n_push = 2 * num_rendered_points;
		pixel_indexer->clear_counters_memory();
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
	}
	if (!settings->fit2D_enable) {
		pixel_indexer->clear_counters_memory();			
		pixel_indexer->assign_pull_constraints_indices(num_sensor_points);
		n_pull = 1 * num_sensor_points; // point to plane
		n_push = 0;
	}

	int n_total = n_pull + n_push;	
      
    // CUDA_TIMED_BLOCK(timer,"memory resize + zero (J+e)")
    { 
        const J_row zeros = {};
		thrust::fill(J->begin(), J->begin() + n_total, zeros);
		thrust::fill(F->begin(), F->begin() + n_total, 0.0f);
        
        if (n_total==0) return;
    }
    J_row* J_push = thrust::raw_pointer_cast(J->data());
    J_row* J_pull = J_push + n_push;    
    
	float* F_push = thrust::raw_pointer_cast(F->data());
    float* F_pull = F_push + n_push;

	ComputeJacobianSilhouette functor_push(J_push, F_push);
	ComputeJacobianData functor_data_model(J_pull, F_pull, reweight);

	int factor = 6;
    if(store_corresps) {
		hmodel_correspondences = new thrust::device_vector<float>(n_pull * factor, -111);
		functor_data_model.store_data(thrust::raw_pointer_cast(hmodel_correspondences->data()));
    }
    
	thrust::sequence(push_indices->begin(), push_indices->begin() + num_rendered_points);

    //CUDA_TIMED_BLOCK(timer,"Assemble Jacobian")
	{
		if (settings->fit2D_enable) {
			//thrust::for_each(_rendered_indicator->begin(), _rendered_indicator->begin() + num_rendered_points, functor_push);
			thrust::for_each(push_indices->begin(), push_indices->begin() + num_rendered_points, functor_push);
		}
		if (settings->fit3D_enable) 		
			thrust::for_each(_sensor_indicator->begin(), _sensor_indicator->begin() + num_sensor_points, functor_data_model);				
	} 

    // CUDA_TIMED_BLOCK(timer, "Jt*J and Jt*e + CPU Transfer")
	{
		CublasHelper::outer_product_J(*J, *JtJ, n_total, NUM_THETAS);
		CublasHelper::vector_product_J(*J, *F, *JtF, n_total, NUM_THETAS);
		thrust::copy(JtF->begin(), JtF->end(), eigen_JtF);
		thrust::copy(JtJ->begin(), JtJ->end(), eigen_JtJ);
	}

	// Multiply with CPU
	if (_test) {
		thrust::host_vector<float> F_host(n_total);
		thrust::copy(F->begin(), F->begin() + n_total, F_host.begin());
		thrust::host_vector<J_row> J_host(n_total);
		thrust::copy(J->begin(), J->begin() + n_total, J_host.begin());

		for (size_t i = 0; i < NUM_THETAS; i++) {
			double result = 0;
			for (size_t k = 0; k < n_total; k++) {				
				J_row a = J_host[k];
				result = result + (double)a.data[i] * (double)F_host[k];				
			}
			eigen_JtF[i] = result;
		}	
		for (size_t i = 0; i < NUM_THETAS; i++) {
			for (size_t j = 0; j < NUM_THETAS; j++) {
				double result = 0;
				for (size_t k = 0; k < n_total; k++) {
					J_row a = J_host[k];
					result = result + (double)a.data[i] * (double)a.data[j];
				}
				eigen_JtJ[i * NUM_THETAS + j] = result;
			}
		}
    }

    /// Only need evaluate metric on the last iteration
    if (eval_metric) {
		thrust::device_vector<float> f_pull(n_pull);
		thrust::transform(F->begin() + n_push, F->begin() + n_push + n_pull, f_pull.begin(), absolute_value());
		pull_error = thrust::reduce(f_pull.begin(), f_pull.end());
		pull_error = pull_error / n_pull;
		//std::cout << pull_error << std::endl;

		thrust::device_vector<float> f_push(n_push);
		thrust::transform(F->begin(), F->begin() + n_push, f_push.begin(), absolute_value());
		push_error = thrust::reduce(f_push.begin(), f_push.end());
		push_error = push_error / n_push;
	}		
	
	//Write the correspondences	
	if (store_corresps) {
		std::ofstream output_file;
		std::string data_path = "...";
		thrust::host_vector<float> output(n_pull * factor);
		
		thrust::copy(hmodel_correspondences->begin(), hmodel_correspondences->begin() + n_pull * factor, output.begin());
		output_file.open(data_path + "corresp-" + std::to_string(id) + ".txt");
		for (size_t i = 0; i < n_pull * factor; i++) {
			output_file << output[i] << " ";
		}
		output_file.close();
		
	}	
    return;
}
